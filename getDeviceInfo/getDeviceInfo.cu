#include <stdio.h>
#include <iostream>

int main() {
  int nDevices;

  hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) {
      std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
      return 1;
  }

  std::cout << "Number of devices: " << nDevices << std::endl;
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipError_t err = hipGetDeviceProperties(&prop, i);
    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    std::cout << "Device Number: " << i
        << "\n  Device name: " << prop.name
        << "\n  Memory Clock Rate (KHz): " << prop.memoryClockRate
        << "\n  Memory Bus Width (bits): " << prop.memoryBusWidth
        << "\n  Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 
        << std::endl;
  }

  return 0;
}
